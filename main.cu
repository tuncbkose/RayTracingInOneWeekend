#include "hip/hip_runtime.h"
#include <iostream>

#include "geometry/hittable.h"
#include "geometry/hittable_list.h"
#include "geometry/sphere.h"
#include "utils/common.h"
#include "utils/cuda_utils.h"


__device__ vec3 ray_color(const ray& r, hittable** world){
    /*
     * Calculate the color of given ray
     * 'f's enforce single precision arithmetic for GPU performance
     */
    hit_record rec;
    if ((*world)->hit(r, 0, infinity, rec)) return 0.5f*(rec.normal + vec3(1,1,1));
    
    vec3 unit_dir = unit_vector(r.direction());
    auto a = 0.5f*(unit_dir.y() + 1.f);
    return (1.f-a)*vec3(1., 1., 1.) + a*vec3(0.5, 0.7, 1.);
}

__global__ void render(vec3* fb, int max_x, int max_y, vec3 top_left_loc, 
                        vec3 delta_horizontal, vec3 delta_vertical, vec3 origin, hittable** world){
    /* 
     * - top_left_loc: location of the top-left pixel in the image
     * - delta_horizontal: horizontal difference between pixels (going left)
     * - delta_vertical : vertical difference between pixels (going down)
     * - origin: location of camera
     */
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // if out of bounds, do nothing
    if ((ix >= max_x) || (iy >= max_y)) return;

    auto pixel_center = top_left_loc + (ix*delta_horizontal) + (iy*delta_vertical);
    ray r(origin, pixel_center-origin);
    
    fb[ix + iy*max_x] = ray_color(r, world);
}

__global__ void create_world(hittable** d_list, hittable** d_world){
    /*
     * Create the objects in the GPU
     */
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *(d_list) = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_list, 2);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world){
    /* 
     *  Free GPU allocated memory
     *  the warnings are fine since these are always a derived class
     */
    if (threadIdx.x == 0 && blockIdx.x == 0){
        delete *(d_list);
        delete *(d_list+1);
        delete *(d_world);
    }
}

int main(){

    // Image

    auto aspect_ratio = 16. / 9.;
    int image_width = 400;
    int image_height = static_cast<int>(image_width / aspect_ratio);
    image_height = (image_height >= 1) ? image_height : 1;

    // Camera
    auto viewport_height = 2.;
    auto viewport_width = viewport_height * (static_cast<double>(image_width)/image_height);
    auto focal_length = 1.;
    auto camera_center = vec3(0, 0, 0);
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // horizontal/vertical dist between pixels in the viewport
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Figure out location of top left pixel
    auto viewport_upper_left = camera_center - vec3(0,0, focal_length) - viewport_u/2 - viewport_v/2;
    auto pixel00_loc = viewport_upper_left + 0.5*(pixel_delta_u + pixel_delta_v);
            
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels*sizeof(vec3);

    // Initialize the world
    hittable** d_list;
    checkCudaErrors(hipMalloc(&d_list, 2*sizeof(hittable*))); // clangd warns this but this is fine
    hittable** d_world;
    checkCudaErrors(hipMalloc(&d_world, sizeof(hittable*)));
    create_world<<<1,1>>>(d_list, d_world);  // 1,1 because we need to do this only once
    checkCudaErrors(hipGetLastError());
    
    // Determine blocks and threads for CUDA
    int tx = 8;
    int ty = 8;
    dim3 blocks(divup(image_width, tx), divup(image_height, ty));
    dim3 threads(tx, ty);  // should be multiple of 32

    // Render

    // Allocate frame buffer in GPU
    vec3* fb_gpu;
    checkCudaErrors(hipMalloc(&fb_gpu, fb_size));

    render<<<blocks, threads>>>(fb_gpu, image_width, image_height, 
        pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center, d_world);
    checkCudaErrors(hipGetLastError());

    // Copy results to host
    vec3* fb_host = new vec3[num_pixels];
    checkCudaErrors(hipMemcpy(fb_host, fb_gpu, fb_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(fb_gpu));  // we are done with GPU memory
    free_world<<<1,1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    
    // Output to file
    
    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; ++j) {
        for (int i = 0; i < image_width; ++i) {
            auto pixel = fb_host[i + j*image_width];
            int ir = static_cast<int>(255.999 * pixel.x());
            int ig = static_cast<int>(255.999 * pixel.y());
            int ib = static_cast<int>(255.999 * pixel.z());

            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }

    delete[] fb_host;
}
